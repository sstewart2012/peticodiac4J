
#include <hip/hip_runtime.h>
#define OFFSET(row, col, ncols) (row * ncols + col)
#define NO_BOUND -1
#define EPSILON 0.000001
#define NONBASIC_FLAG 0
#define BASIC_FLAG 1

extern "C"
__global__ void check_bounds(
	const int n,
	const int offset,
	const float* const lower,
	const float* const upper,
	const float* const assigns,
	const unsigned char* const flags,
	int* const result
)
{
	const int idx = offset + blockIdx.x * blockDim.x + threadIdx.x;

	// Boundary check and nonbasic variables are skipped
	if (idx >= n || flags[idx] == NONBASIC_FLAG)
		return;

	//printf("[%d] n=%d offset=%d flags=%d\n", idx, n, offset, flags[idx]); return;

	//printf("[%d] lowPtr=%p uppPtr=%p offset=%d n=%d flagPtr=%p\n", idx, lower, upper, *offset, n, flags);
	const float ass = assigns[idx];
	const float low = lower[idx];
	const float upp = upper[idx];
	//printf("[%d] low=%f ass=%f upp=%f\n", idx, low, ass, upp);

	const bool testA = fabsf(ass - low) < EPSILON;
	const bool testB = fabsf(ass - upp) < EPSILON;
	const bool testC = low != NO_BOUND && ass < low;
	const bool testD = upp != NO_BOUND && ass > upp;

	if (testA || testB || !(testC || testD)) {
		return;
	} else {
		atomicMin(result, idx);
		//printf("Variable %d is broken (result=%d).\n", idx, *result);
	}
}

#define NONE_FOUND -1
#define IS_INCREASABLE(low, upp, ass) (upp == NO_BOUND || ass < upp)
#define IS_DECREASABLE(low, upp, ass) (low == NO_BOUND || ass > low)

/**
 * If found, returns the index of a suitable variable; otherwise, returns
 * NONE_FOUND. The return value is stored in the output argument called
 * suitable_idx.
 */
 extern "C"
__global__ void find_suitable(
	const int ncols,
	const int broken_idx,
	const int offset,
	const float* const tableau,
	const float* const lower,
	const float* const upper,
	const float* const assigns,
	const unsigned char* const flags,
	const int* const varToTableau,
	const int* const colToVar,
	int* const suitable_idx
){
	// Determine variable index assigned to this thread
	const int idx = offset + (blockIdx.x * blockDim.x + threadIdx.x);
	const int var = colToVar[idx];

	// Boundary check and "basic" variables are skipped
	if (idx >= ncols || flags[var] == BASIC_FLAG)
		return;

	// Determine if the broken variable needs to be increased or decreased
	const bool increase = assigns[broken_idx] < lower[broken_idx];

	// Read bounds information needed to determine if potential suitable variable
	// is increaseable or decreaseable
	const float ass = assigns[var];
	const float low = lower[var];
	const float upp = upper[var];

	// Obtain coefficient value in the tableau
	const float coeff = tableau[varToTableau[broken_idx] * ncols + varToTableau[var]];

	//printf("[%d] offset=%d ncols=%d low=%f ass=%f upp=%f increase=%d coeff=%f\n",
	//	idx, offset, ncols, low, ass, upp, increase, coeff);
	
	if (increase){
		if ((IS_INCREASABLE(low, upp, ass) && coeff > 0) 
				|| (IS_DECREASABLE(low, upp, ass) && coeff < 0)) {
	        atomicMin(suitable_idx, var);
	        //printf("Variable %d is suitable\n", idx, suitable_idx);
		}
	}
	else {
		if ((IS_INCREASABLE(low, upp, ass) && coeff < 0) 
				|| (IS_DECREASABLE(low, upp, ass) && coeff > 0)) {
	        atomicMin(suitable_idx, var);
	        //printf("Variable %d is suitable\n", idx, suitable_idx);
		}
	}
}

extern "C"
__global__ void find_suitable_complete(
	const int ncols,
	const int broken_idx,
	const int suitable_idx,
	const float* const tableau,
	const float* const lower,
	const float* const upper,
	float* const assigns,
	const int* const varToTableau
){
	if (blockIdx.x * blockDim.x + threadIdx.x > 0)
		return;

	// Read bounds information for the broken variable
	float ass = assigns[broken_idx];
	float low = lower[broken_idx];
	float upp = upper[broken_idx];

	// Determine if the broken variable needs to be increased or decreased
	const bool increase = ass < low;

	// Obtain coefficient value in the tableau
	const float coeff = tableau[varToTableau[broken_idx] * ncols
		+ varToTableau[suitable_idx]];

	// Amounts to adjust assignments of suitable and broken variables
	const float delta = increase ? low - ass : ass - upp;
	const float theta = delta / coeff;

	//printf("[%d] b=%d s=%d increase=%d delta=%f theta=%f\n",
	//	threadIdx.x, broken_idx, suitable_idx, increase, delta, theta);

	// Read bounds info for the suitable variable to check if
	// increaseable or decreaseable
	ass = assigns[suitable_idx];
	low = lower[suitable_idx];
	upp = upper[suitable_idx];

	if (increase) {
		if ((IS_INCREASABLE(low, upp, ass) && coeff > 0) ||
				(IS_DECREASABLE(low, upp, ass) && coeff < 0)) {
			assigns[suitable_idx] += coeff < 0 ? -theta : theta;
			assigns[broken_idx] += delta;
			//printf("a(%d) = %f\n", broken_idx, assigns[broken_idx]);
			//printf("a(%d) = %f\n", suitable_idx, assigns[suitable_idx]);
		}
	}
	else {
		if ((IS_INCREASABLE(low, upp, ass) && coeff < 0) ||
				(IS_DECREASABLE(low, upp, ass) && coeff > 0)) {
			assigns[suitable_idx] -= coeff < 0 ? theta : -theta;
			assigns[broken_idx] -= delta;
			//printf("a(%d) = %f\n", broken_idx, assigns[broken_idx]);
			//printf("a(%d) = %f\n", suitable_idx, assigns[suitable_idx]);
		}
	}
}

extern "C"
__global__ void pivot_update_inner(
	const float alpha,
	const int pivot_row,
	const int pivot_col,
	const int nrows,
	const int ncols,
	float* const tableau
){
	// Determine thread ID in 2D (x and y)
	const unsigned int col = blockDim.x * blockIdx.x + threadIdx.x; // column index
	const unsigned int row = blockDim.y * blockIdx.y + threadIdx.y; // row index

	if (col < ncols && row < nrows && row != pivot_row && col != pivot_col) {
		// Compute helpful indices
		const unsigned int delta_row_idx = OFFSET(row, 0, ncols);
		const unsigned int delta_idx = delta_row_idx + col;

		// Load values from global memory
		const float delta = tableau[delta_idx];
		const float beta = tableau[OFFSET(pivot_row, col, ncols)];
		const float gamma = tableau[delta_row_idx + pivot_col];

		// Store result
		float coeff = delta - (beta * gamma) / alpha;
		tableau[delta_idx] = coeff;
	}
}

extern "C"
__global__ void pivot_update_row(
	const float alpha,
	const int row,
	const int ncols,
	float* const tableau
){
	float* const tableau_row = &tableau[row * ncols];
	const int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (col >= ncols)
		return;
	const float beta = tableau_row[col];
	const float coeff = -beta / alpha;
	tableau_row[col] = coeff;
}

extern "C"
__global__ void pivot_update_column(
	const float alpha,
	const int col,
	const int nrows,
	const int ncols,
	float* const tableau
){
	float* const tableau_col = tableau + col;
	const int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row >= nrows)
		return;
	const int idx = row * ncols;
	const float gamma = tableau_col[idx];
	tableau_col[idx] = gamma / alpha;
}

extern "C"
__global__ void update_assignment(
	const int n,
	const float* const input,
	const float* const assigns,
	const int* const colToVar,
	float* const output
){
	extern __shared__ float partial_sums[];
	const int gid = blockDim.x * blockIdx.x + threadIdx.x;
	const int lid = threadIdx.x;
	
	// Boundary check
	if (gid >= n)
		return;

	//printf("[%d] n=%d\n", gid, n); return;

	// Pre-fetch and multiply by corresponding assignment
	const float a = assigns[colToVar[gid]];
	partial_sums[lid] = a * input[gid];
	__syncthreads();

	//printf("[%d] n=%d psum=%f\n", gid, n, partial_sums[lid]);
	//return;

	// Reduce using interleaved pairs
	for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
		if (lid < stride) {
			partial_sums[lid] += partial_sums[lid + stride];
		}
		__syncthreads();
	}

	// Write the result for this block to global memory
	if (lid == 0) {
		output[blockIdx.x] = partial_sums[0];
	}
}

extern "C"
__global__ void update_assignment_complete(
	const int var,
	const float* const input,
	float* const assigns
){
	extern __shared__ float partial_sums[];
	const int lid = threadIdx.x;
	//printf("[%d] var=%d input=%f\n", lid, var, input[idx], input);

	// Pre-fetch
	partial_sums[lid] = input[lid];
	__syncthreads();
	//printf("[%d] offset=%d var=%d partial_sums=%f\n", lid, offset, var, partial_sums[idx]);

	// Reduce
	for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
		if (lid < stride) {
			partial_sums[lid] += partial_sums[lid + stride];
		}
		__syncthreads();
	}

	// Write the result to the assignments array
	if (lid == 0) {
		assigns[var] = partial_sums[0];
		//printf("[%d] a(%d)=%f\n", lid, var, assigns[var]);
	}
}